#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *//*
 */

/** @file   optimizer.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  API interface of optimizers that can be used to train models.
 */

#include <tiny-cuda-nn/optimizer.h>

#include <tiny-cuda-nn/optimizers/adam.h>
#include <tiny-cuda-nn/optimizers/average.h>
#include <tiny-cuda-nn/optimizers/batched.h>
#include <tiny-cuda-nn/optimizers/ema.h>
#include <tiny-cuda-nn/optimizers/composite.h>
#include <tiny-cuda-nn/optimizers/exponential_decay.h>
#include <tiny-cuda-nn/optimizers/lookahead.h>
#include <tiny-cuda-nn/optimizers/novograd.h>
#include <tiny-cuda-nn/optimizers/sgd.h>

#ifdef TCNN_SHAMPOO
#include <tiny-cuda-nn/optimizers/shampoo.h>
#endif


TCNN_NAMESPACE_BEGIN

template <typename T>
Optimizer<T>* create_optimizer(const json& optimizer) {
	std::string optimizer_type = optimizer.value("otype", "Adam");

	if (equals_case_insensitive(optimizer_type, "Adam")) {
		return new AdamOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "Average")) {
		return new AverageOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "Batched")) {
		return new BatchedOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "Composite")) {
		return new CompositeOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "Ema")) {
		return new EmaOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "ExponentialDecay")) {
		return new ExponentialDecayOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "Lookahead")) {
		return new LookaheadOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "Novograd")) {
		return new NovogradOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "SGD")) {
		return new SGDOptimizer<T>{optimizer};
	} else if (equals_case_insensitive(optimizer_type, "Shampoo")) {
#ifdef TCNN_SHAMPOO
		return new ShampooOptimizer<T>{optimizer};
#else
		throw std::runtime_error{"The Shampoo optimizer is only available when compiling with CUDA 11 or higher."};
#endif
	} else {
		throw std::runtime_error{fmt::format("Invalid optimizer type: {}", optimizer_type)};
	}
}

template Optimizer<float>* create_optimizer(const json& optimizer);
template Optimizer<__half>* create_optimizer(const json& optimizer);

TCNN_NAMESPACE_END
