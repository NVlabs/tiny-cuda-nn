#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/** @file   network.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  API interface of a neural network implementation
 */

#include <tiny-cuda-nn/common_device.h>
#include <tiny-cuda-nn/network.h>

#include <tiny-cuda-nn/networks/cutlass_mlp.h>

#if TCNN_MIN_GPU_ARCH > 70
#include <tiny-cuda-nn/networks/fully_fused_mlp.h>
#endif

TCNN_NAMESPACE_BEGIN

Activation string_to_activation(const std::string& activation_name) {
	if (equals_case_insensitive(activation_name, "None")) {
		return Activation::None;
	} else if (equals_case_insensitive(activation_name, "ReLU")) {
		return Activation::ReLU;
	} else if (equals_case_insensitive(activation_name, "Exponential")) {
		return Activation::Exponential;
	} else if (equals_case_insensitive(activation_name, "Sigmoid")) {
		return Activation::Sigmoid;
	} else if (equals_case_insensitive(activation_name, "Sine")) {
		return Activation::Sine;
	} else if (equals_case_insensitive(activation_name, "Squareplus")) {
		return Activation::Squareplus;
	} else if (equals_case_insensitive(activation_name, "Softplus")) {
		return Activation::Softplus;
	}

	throw std::runtime_error{fmt::format("Invalid activation name: {}", activation_name)};
}

std::string to_string(Activation activation) {
	switch (activation) {
		case Activation::None: return "None";
		case Activation::ReLU: return "ReLU";
		case Activation::Exponential: return "Exponential";
		case Activation::Sigmoid: return "Sigmoid";
		case Activation::Sine: return "Sine";
		case Activation::Squareplus: return "Squareplus";
		case Activation::Softplus: return "Softplus";
		default: throw std::runtime_error{"Invalid activation."};
	}
}

template <typename T>
void extract_dimension_pos_neg(hipStream_t stream, const uint32_t num_elements, const uint32_t dim, const uint32_t fan_in, const uint32_t fan_out, const T* encoded, MatrixLayout layout, float* output) {
	linear_kernel(extract_dimension_pos_neg_kernel<T>, 0, stream, num_elements, dim, fan_in, fan_out, encoded, layout, output);
}

template void extract_dimension_pos_neg(hipStream_t stream, const uint32_t num_elements, const uint32_t dim, const uint32_t fan_in, const uint32_t fan_out, const network_precision_t* encoded, MatrixLayout layout, float* output);

std::string select_network(const json& network) {
	std::string otype = network.value("otype", "MLP");
	bool want_fully_fused_mlp = equals_case_insensitive(otype, "MegakernelMLP") || equals_case_insensitive(otype, "FullyFusedMLP");
	bool want_cutlass_mlp = equals_case_insensitive(otype, "MLP") || equals_case_insensitive(otype, "CutlassMLP");

	// If the GPU architecture is insufficient for
	if (MIN_GPU_ARCH <= 70 || std::is_same<network_precision_t, float>::value) {
		if (want_fully_fused_mlp && MIN_GPU_ARCH <= 70) {
			std::cout
				<< "Warning: FullyFusedMLP is not supported for the selected architecture " << MIN_GPU_ARCH << ". "
				<< "Falling back to CutlassMLP. For maximum performance, raise the target GPU architecture to 75+."
				<< std::endl;
		}

		want_cutlass_mlp |= want_fully_fused_mlp;
		want_fully_fused_mlp = false;
	}

	if (want_fully_fused_mlp) {
		return "FullyFusedMLP";
	} else if (want_cutlass_mlp) {
		return "CutlassMLP";
	} else {
		return otype;
	}
}

uint32_t minimum_alignment(const json& network) {
	std::string network_type = select_network(network);

	if (equals_case_insensitive(network_type, "FullyFusedMLP")) {
#if TCNN_MIN_GPU_ARCH > 70
		uint32_t n_neurons = network.value("n_neurons", 128u);
		switch (n_neurons) {
			case  16: return FullyFusedMLP<network_precision_t,  16>::REQUIRED_ALIGNMENT();
			case  32: return FullyFusedMLP<network_precision_t,  32>::REQUIRED_ALIGNMENT();
			case  64: return FullyFusedMLP<network_precision_t,  64>::REQUIRED_ALIGNMENT();
			case 128: return FullyFusedMLP<network_precision_t, 128>::REQUIRED_ALIGNMENT();
			default: throw std::runtime_error{fmt::format("FullyFusedMLP only supports 16, 32, 64, and 128 neurons, but got {}. Use CutlassMLP instead if this is a requirement.", n_neurons)};
		}
#else
		throw std::runtime_error{"FullyFusedMLP was not compiled due to insufficient GPU arch of <70."};
#endif
	} else {
		return CutlassMLP<network_precision_t>::REQUIRED_ALIGNMENT();
	}
}

template <typename T>
Network<T>* create_network(const json& network) {
	std::string network_type = select_network(network);

	if (equals_case_insensitive(network_type, "FullyFusedMLP")) {
		if (!std::is_same<network_precision_t, __half>::value) {
			throw std::runtime_error{"FullyFusedMLP can only be used if the network precision is set to __half."};
		} else {
#if TCNN_MIN_GPU_ARCH > 70
#  define TCNN_FULLY_FUSED_PARAMS \
	network["n_input_dims"], \
	network["n_output_dims"], \
	network.value("n_hidden_layers", 5u), \
	string_to_activation(network.value("activation", "ReLU")), \
	string_to_activation(network.value("output_activation", "None")),

			uint32_t n_neurons = network.value("n_neurons", 128u);
			switch (n_neurons) {
				case  16: return new FullyFusedMLP<T,  16>{TCNN_FULLY_FUSED_PARAMS};
				case  32: return new FullyFusedMLP<T,  32>{TCNN_FULLY_FUSED_PARAMS};
				case  64: return new FullyFusedMLP<T,  64>{TCNN_FULLY_FUSED_PARAMS};
				case 128: return new FullyFusedMLP<T, 128>{TCNN_FULLY_FUSED_PARAMS};
				default: throw std::runtime_error{fmt::format("FullyFusedMLP only supports 16, 32, 64, and 128 neurons, but got {}. Use CutlassMLP instead if this is a requirement.", n_neurons)};
			}
#  undef TCNN_FULLY_FUSED_PARAMS
#else //TCNN_MIN_GPU_ARCH > 70
			throw std::runtime_error{"FullyFusedMLP was not compiled due to insufficient GPU arch of <=70."};
#endif //TCNN_MIN_GPU_ARCH > 70
		}
	} else if (equals_case_insensitive(network_type, "CutlassMLP")) {
		return new CutlassMLP<T>{
			network["n_input_dims"],
			network.value("n_neurons", 128u),
			network["n_output_dims"],
			network.value("n_hidden_layers", 5u),
			string_to_activation(network.value("activation", "ReLU")),
			string_to_activation(network.value("output_activation", "None")),
		};
	}

	throw std::runtime_error{fmt::format("Invalid network type: {}", network_type)};
}

template Network<network_precision_t>* create_network(const json& network);

TCNN_NAMESPACE_END
